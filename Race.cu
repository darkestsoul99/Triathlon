#include "hip/hip_runtime.h"
#include "Race.cuh"
#include "Athlete.cuh"
#include <iostream>
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""
#include <thread>
#include <chrono>

#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// CUDA kernel to update athlete positions
__global__ void updatePositions(Athlete* athletes, float raceTime) {
    int segment_distances[3] = { 5000, 45000, 100000 }; // Swimming, Cycling, Running distances
    int num_athletes = 900;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < num_athletes; i += stride) {
        if (athletes[i].race_finished == false) {
            // Update athlete's position
            athletes[i].position += athletes[i].speed;
            athletes[i].time += 1; // 1 second per update

            // Handle segment transitions
            if (athletes[i].segment == 0 && athletes[i].position >= segment_distances[0]) {
                athletes[i].speed *= 3;
                athletes[i].time += 10;
                athletes[i].segment = 1;
                athletes[i].position = segment_distances[0]; // Exact segment boundary
            }
            else if (athletes[i].segment == 1 && athletes[i].position >= segment_distances[1]) {
                athletes[i].speed /= 3;
                athletes[i].time += 10;
                athletes[i].segment = 2;
                athletes[i].position = segment_distances[1]; // Exact segment boundary
            }
            else if (athletes[i].segment == 2 && athletes[i].position >= segment_distances[2]) {
                athletes[i].time += raceTime;
                athletes[i].position = segment_distances[2];
                athletes[i].race_finished = true;
            }
        }
    }
}

Race::Race(int n, std::vector<std::vector<float>>& athlete_speeds) : num_teams(n), raceTime(0.0) {
    for (int i = 0; i < num_teams; ++i) {
        teams.emplace_back(i, athlete_speeds[i].data());
    }
    std::cout << "Race created." << std::endl;
    std::cout << "Number of teams: " << teams.size() << std::endl;
}

hipError_t Race::startRace(const int team_index, const int athlete_index) {
    std::cout << "Race started." << std::endl;
    int num_athletes = num_teams * 3;
    Athlete* athletes;
    int segment_distances[3] = { 5000, 45000, 55000 }; // Swimming, Cycling, Running distances
    hipError_t cudaStatus;
    bool firstAthlete = false;

    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }
    // Allocate managed memory for athletes
    gpuErrchk(cudaStatus = hipMallocManaged(&athletes, num_athletes * sizeof(Athlete)));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMallocManaged failed!");
        goto Error;
    }

    // Initialize athlete data
    for (int i = 0; i < num_teams; ++i) {
        for (int j = 0; j < 3; ++j) {
            int idx = i * 3 + j;
            athletes[idx].team_id = i;
            athletes[idx].speed = teams[i].athletes[j].speed;
            athletes[idx].position = teams[i].athletes[j].position; // Start at the beginning
            athletes[idx].time = teams[i].athletes[j].time; // Start time
            athletes[idx].segment = teams[i].athletes[j].segment; // Start in swimming segment
        }
    }

    while (true) {
        int num_blocks = static_cast<int>((num_athletes + 255) / 256);
        int blockSize = 256;
        updatePositions<<<num_blocks, blockSize>>>(athletes, raceTime);
        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "updatePositions launch failed: %s\n", hipGetErrorString(cudaStatus));
            goto Error;
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching updatePositions!\n", cudaStatus);
            goto Error;
        }

        // Check for race completion
        bool race_ongoing = false;
        for (int i = 0; i < num_athletes; ++i) {
            if (athletes[i].position < segment_distances[2]) {
                race_ongoing = true;
                break;
            }
            else {
                if (firstAthlete == false) {
                    printAthleteResults(athletes);
                    firstAthlete = true;
                }
            }
        }

        // Sleep for a second to simulate real-time updates (if running on a system where this is feasible)
        std::this_thread::sleep_for(std::chrono::seconds(1));
        raceTime++;
        // Print debugging information
        int i = team_index * 3 + athlete_index;
        printf("Athlete %d: Position = %f, Speed = %f, Time = %f, Segment = %d\n", i, athletes[i].position, athletes[i].speed, athletes[i].time, athletes[i].segment);
        if (!race_ongoing) break;
    }

    // Print final results
    printTeamResults(athletes);

Error:
    // Free managed memory
    gpuErrchk(hipFree(athletes));

    return cudaStatus;
}

void Race::printAthleteResults(Athlete* athletes) {
    std::cout << "First Winner has finished the triathlon. Current results :\n";
    // Print individual athlete results
    for (int i = 0; i < num_teams; ++i) {
        for (int j = 0; j < 3; ++j) {
            int idx = i * 3 + j;
            std::cout << "Athlete in team " << teams[i].team_id << " - Position: " << athletes[idx].position
                << ", Speed: " << athletes[idx].speed << ", Time: " << athletes[idx].time << " seconds\n";
        }
    }
}

void Race::printTeamResults(Athlete* athletes) {
    // Calculate team rankings based on total time
    std::vector<std::pair<int, float>> team_times;
    for (int i = 0; i < num_teams; ++i) {
        float total_time = 0.0f;
        for (int j = 0; j < 3; ++j) {
            int idx = i * 3 + j;
            total_time += athletes[idx].time;
        }
        team_times.emplace_back(i, total_time);
    }

    // Sort teams by total time
    std::sort(team_times.begin(), team_times.end(), [](const auto& left, const auto& right) {
        return left.second < right.second;
        });

    // Print team rankings
    std::cout << "Team Rankings:\n";
    for (size_t i = 0; i < team_times.size(); ++i) {
        std::cout << "Rank " << (i + 1) << ": Team " << team_times[i].first
            << " Total Time: " << team_times[i].second << " seconds\n";
    }
}
